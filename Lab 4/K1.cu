#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define BLOCK_SIZE 16

typedef long long ll;

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <filesystem>
#include "stb/stb_image.h"
#include "stb/stb_image_write.h"
using namespace std;

// Kernel function for 3D convolution
__global__ void basicConvolutionKernel(const unsigned char *inputImages, unsigned char *outputImages, const float *mask, int width, int height, int channels, int maskSize, int batchSize) {
    int outCol = blockIdx.x * blockDim.x + threadIdx.x;
    int outRow = blockIdx.y * blockDim.y + threadIdx.y;
    int batchIndex = blockIdx.z * blockDim.z + threadIdx.z;

    if (outCol < width && outRow < height && batchIndex < batchSize) {
        int maskRadius = maskSize / 2;
        float sum = 0.0;
        for (int maskRow = 0; maskRow < maskSize; maskRow++) {
            for (int maskCol = 0; maskCol < maskSize; maskCol++) {
                int inRow = outRow - maskRadius + maskRow;
                int inCol = outCol - maskRadius + maskCol;
                if (inRow >= 0 && inRow < height && inCol >= 0 && inCol < width) {
                    for (int c = 0; c < channels; c++) {
                        sum += mask[maskRow * maskSize + maskCol] * (float)inputImages[batchIndex * width * height * channels + inRow * width * channels + inCol * channels + c];
                    }
                }
            }
        }
        sum = sum < 0 ? 0 : sum;
        sum = sum > 255 ? 255 : sum;
        outputImages[batchIndex * width * height + outRow * width + outCol] = (unsigned char) sum;
    }
}

int main(int argc, char **argv) {
    // Check for the correct number of arguments
    if (argc != 5) {
        printf("Usage: %s input_folder output_folder batch_size mask_file\n", argv[0]);
        return 1;
    }

    // Read command line arguments
    string inputFolder = argv[1];
    string outputFolder = argv[2];
    int batchSize = atoi(argv[3]);
    char* maskFile = argv[4];

    // Print the command line arguments
    printf("Input folder: %s\n", inputFolder.c_str());
    printf("Output folder: %s\n", outputFolder.c_str());
    printf("Batch size: %d\n", batchSize);
    printf("Mask file: %s\n", maskFile);
    printf("---------------------------------------------\n");

    // Check if the input folder exists
    if (!filesystem::exists(inputFolder)) {
        printf("Error: Input folder does not exist\n");
        return 1;
    }

    // Check if the mask file exists
    if (!filesystem::exists(maskFile)) {
        printf("Error: Mask file does not exist\n");
        return 1;
    }

    // Find the number of images in the input folder
    int numImages = 0;
    for (const auto &entry: filesystem::directory_iterator(inputFolder)) numImages++;
    printf("Number of images: %d\n", numImages);

    // Check if the number of images is less than the batch size
    if (numImages < batchSize) {
        printf("Error: Number of images is less than the batch size\n");
        return 1;
    }

    // Get the width, height, and number of channels of the first image in the input folder
    int width, height, channels;
    string firstImage = filesystem::directory_iterator(inputFolder)->path().string();
    unsigned char *image = stbi_load(firstImage.c_str(), &width, &height, &channels, STBI_rgb);
    if (image == NULL) {
        printf("Error: Failed to load image\n");
        return 1;
    }

    // Print the width, height, and number of channels of the image
    printf("Image width: %d\n", width);
    printf("Image height: %d\n", height);
    printf("Number of channels: %d\n", channels);

    // Calculate the size of the images
    int imgSize = width * height * channels;
    printf("Image size: %d\n", imgSize);
    printf("---------------------------------------------\n");

    // Allocate memory for input and output images
    unsigned char *inputImages = (unsigned char *)malloc(imgSize * batchSize * sizeof(unsigned char));
    unsigned char *outputImages = (unsigned char *)malloc(imgSize * batchSize * sizeof(unsigned char));

    // Read input images from the input folder and copy them to the inputImages array
    int i = 0;
    for (const auto &entry: filesystem::directory_iterator(inputFolder)) {
        string imagePath = entry.path().string();
        unsigned char *image = stbi_load(imagePath.c_str(), &width, &height, &channels, STBI_rgb);
        if (image == NULL) {
            printf("Error: Failed to load image\n");
            return 1;
        }
        memcpy(inputImages + i * imgSize, image, imgSize * sizeof(unsigned char));
        stbi_image_free(image);
        i++;
        if (i == batchSize) break;
    }
    
    // Read convolution mask from the provided text file
    FILE *fp = fopen(maskFile, "r");
    if (fp == NULL) {
        printf("Error: Unable to open mask file\n");
        return 1;
    }
    int maskSize;
    fscanf(fp, "%d", &maskSize);
    printf("Mask size: %d\n", maskSize);
    float *mask = (float *)malloc(maskSize * maskSize * sizeof(float));
    if (mask == NULL) {
        printf("Error: Memory allocation failed for mask\n");
        fclose(fp);
        return 1;
    }
    for (int i = 0; i < maskSize * maskSize; i++) {
        fscanf(fp, "%f", &mask[i]);
    }
    fclose(fp);

    // Print the mask
    printf("Mask:\n");
    for (int i = 0; i < maskSize; i++) {
        for (int j = 0; j < maskSize; j++) {
            printf("%f ", mask[i * maskSize + j]);
        }
        printf("\n");
    }
    printf("---------------------------------------------\n");

    // Allocate memory on GPU for input images, output images, and mask
    unsigned char *d_inputImages, *d_outputImages;
    float *d_mask;
    hipMalloc(&d_inputImages, imgSize * batchSize * sizeof(unsigned char));
    hipMalloc(&d_outputImages, imgSize * batchSize * sizeof(unsigned char));
    hipMalloc(&d_mask, maskSize * maskSize * sizeof(float));

    // Copy input images and mask from host to device
    hipMemcpy(d_mask, mask, maskSize * maskSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_inputImages, inputImages, imgSize * batchSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Calculate grid and block dimensions
    dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y, (batchSize + block.z - 1) / block.z);

    // Launch kernel for all images in the batch
    basicConvolutionKernel<<<grid, block>>>(d_inputImages, d_outputImages, d_mask, width, height, channels, maskSize, batchSize);

    // Copy results (output images) from device to host
    hipMemcpy(outputImages, d_outputImages, imgSize * batchSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Create output directory if it does not exist
    if (!filesystem::exists(outputFolder)) {
        filesystem::create_directory(outputFolder);
    }

    // Save output images to the specified output folder
    for (int i = 0; i < batchSize; i++) {
        string outputImagePath = outputFolder + "/" + to_string(i) + ".jpg";
        stbi_write_jpg(outputImagePath.c_str(), width, height, 1, outputImages + i * width * height, width);
    }
    printf("Output images saved to %s\n", outputFolder.c_str());

    // Free memory
    free(inputImages);
    free(outputImages);
    free(mask);
    hipFree(d_inputImages);
    hipFree(d_outputImages);
    hipFree(d_mask);

    return 0;
}
