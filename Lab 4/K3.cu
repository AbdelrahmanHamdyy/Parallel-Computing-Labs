#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define TILE_SIZE 16

typedef long long ll;

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <filesystem>
#include "stb/stb_image.h"
#include "stb/stb_image_write.h"
using namespace std;

// Kernel function for 3D convolution
__global__ void outputTileConvolutionKernel(const unsigned char *inputImages, unsigned char *outputImages, const float *mask, int width, int height, int channels, int maskSize, int batchSize) {
    extern __shared__ float tile[];

    int maskRadius = maskSize / 2;

    int tileDim = TILE_SIZE + maskSize - 1;

    int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int batchIndex = blockIdx.z * blockDim.z + threadIdx.z;

    for (int i = threadIdx.y; i < tileDim; i += blockDim.y) {
        for (int j = threadIdx.x; j < tileDim; j += blockDim.x) {
            int inCol = (blockIdx.x * blockDim.x) - maskRadius + j;
            int inRow = (blockIdx.y * blockDim.y) - maskRadius + i;
            int imgIndex = (batchIndex * height * width + inRow * width + inCol) * channels;
            int tileIndex = i * tileDim + j;
            bool validPixel = (inCol >= 0 && inCol < width && inRow >= 0 && inRow < height);
            tile[tileIndex] = validPixel ? (float)inputImages[imgIndex] + (float)inputImages[imgIndex + 1] + (float)inputImages[imgIndex + 2] : 0;
        }
    }

    __syncthreads();

    if (xIndex < width && yIndex < height && batchIndex < batchSize) {
        float sum = 0.0;
        for (int i = 0; i < maskSize; i++) {
            for (int j = 0; j < maskSize; j++) {
                sum += mask[i * maskSize + j] * tile[(threadIdx.y + i) * tileDim + threadIdx.x + j];
            }
        }
        sum = sum < 0 ? 0 : sum;
        sum = sum > 255 ? 255 : sum;
        outputImages[batchIndex * height * width + yIndex * width + xIndex] = (unsigned char)sum;
    }

    __syncthreads();
}

int main(int argc, char **argv) {
    // Check for the correct number of arguments
    if (argc != 5) {
        printf("Usage: %s input_folder output_folder batch_size mask_file\n", argv[0]);
        return 1;
    }

    // Read command line arguments
    string inputFolder = argv[1];
    string outputFolder = argv[2];
    int batchSize = atoi(argv[3]);
    char* maskFile = argv[4];

    // Print the command line arguments
    printf("Input folder: %s\n", inputFolder.c_str());
    printf("Output folder: %s\n", outputFolder.c_str());
    printf("Batch size: %d\n", batchSize);
    printf("Mask file: %s\n", maskFile);
    printf("---------------------------------------------\n");

    // Check if the input folder exists
    if (!filesystem::exists(inputFolder)) {
        printf("Error: Input folder does not exist\n");
        return 1;
    }

    // Check if the mask file exists
    if (!filesystem::exists(maskFile)) {
        printf("Error: Mask file does not exist\n");
        return 1;
    }

    // Find the number of images in the input folder
    int numImages = 0;
    for (const auto &entry: filesystem::directory_iterator(inputFolder)) numImages++;
    printf("Number of images: %d\n", numImages);

    // Check if the number of images is less than the batch size
    if (numImages < batchSize) {
        printf("Error: Number of images is less than the batch size\n");
        return 1;
    }

    // Get the width, height, and number of channels of the first image in the input folder
    int width, height, channels;
    string firstImage = filesystem::directory_iterator(inputFolder)->path().string();
    unsigned char *image = stbi_load(firstImage.c_str(), &width, &height, &channels, STBI_rgb);
    if (image == NULL) {
        printf("Error: Failed to load image\n");
        return 1;
    }

    // Print the width, height, and number of channels of the image
    printf("Image width: %d\n", width);
    printf("Image height: %d\n", height);
    printf("Number of channels: %d\n", channels);

    // Calculate the size of the images
    int imgSize = width * height * channels;
    printf("Image size: %d\n", imgSize);
    printf("---------------------------------------------\n");

    // Allocate memory for input and output images
    unsigned char *inputImages = (unsigned char *)malloc(imgSize * batchSize * sizeof(unsigned char));
    unsigned char *outputImages = (unsigned char *)malloc(imgSize * batchSize * sizeof(unsigned char));

    // Read input images from the input folder and copy them to the inputImages array
    int i = 0;
    for (const auto &entry: filesystem::directory_iterator(inputFolder)) {
        string imagePath = entry.path().string();
        unsigned char *image = stbi_load(imagePath.c_str(), &width, &height, &channels, STBI_rgb);
        if (image == NULL) {
            printf("Error: Failed to load image\n");
            return 1;
        }
        memcpy(inputImages + i * imgSize, image, imgSize * sizeof(unsigned char));
        stbi_image_free(image);
        i++;
        if (i == batchSize) break;
    }
    
    // Read convolution mask from the provided text file
    FILE *fp = fopen(maskFile, "r");
    if (fp == NULL) {
        printf("Error: Unable to open mask file\n");
        return 1;
    }
    int maskSize;
    fscanf(fp, "%d", &maskSize);
    printf("Mask size: %d\n", maskSize);
    float *mask = (float *)malloc(maskSize * maskSize * sizeof(float));
    if (mask == NULL) {
        printf("Error: Memory allocation failed for mask\n");
        fclose(fp);
        return 1;
    }
    for (int i = 0; i < maskSize * maskSize; i++) {
        fscanf(fp, "%f", &mask[i]);
    }
    fclose(fp);

    // Print the mask
    printf("Mask:\n");
    for (int i = 0; i < maskSize; i++) {
        for (int j = 0; j < maskSize; j++) {
            printf("%f ", mask[i * maskSize + j]);
        }
        printf("\n");
    }
    printf("---------------------------------------------\n");

    // Allocate memory on GPU for input images, output images, and mask
    unsigned char *d_inputImages, *d_outputImages;
    float *d_mask;
    hipMalloc(&d_inputImages, imgSize * batchSize * sizeof(unsigned char));
    hipMalloc(&d_outputImages, imgSize * batchSize * sizeof(unsigned char));
    hipMalloc(&d_mask, maskSize * maskSize * sizeof(float));

    // Copy input images and mask from host to device
    hipMemcpy(d_mask, mask, maskSize * maskSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_inputImages, inputImages, imgSize * batchSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Calculate grid and block dimensions
    dim3 block(TILE_SIZE, TILE_SIZE, 1);
    dim3 grid((width + block.x - 1) / TILE_SIZE, (height + block.y - 1) / TILE_SIZE, (batchSize + block.z - 1) / block.z);

    // Calculate shared memory size
    int sharedMemorySize = pow(TILE_SIZE + maskSize - 1, 2) * sizeof(float);

    // Launch kernel for all images in the batch
    outputTileConvolutionKernel<<<grid, block, sharedMemorySize>>>(d_inputImages, d_outputImages, d_mask, width, height, channels, maskSize, batchSize);

    // Copy results (output images) from device to host
    hipMemcpy(outputImages, d_outputImages, imgSize * batchSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Create output directory if it does not exist
    if (!filesystem::exists(outputFolder)) {
        filesystem::create_directory(outputFolder);
    }

    // Save output images to the specified output folder
    for (int i = 0; i < batchSize; i++) {
        string outputImagePath = outputFolder + "/" + to_string(i) + ".jpg";
        stbi_write_jpg(outputImagePath.c_str(), width, height, 1, outputImages + i * width * height, width);
    }
    printf("Output images saved to %s\n", outputFolder.c_str());

    // Free memory
    free(inputImages);
    free(outputImages);
    free(mask);
    hipFree(d_inputImages);
    hipFree(d_outputImages);
    hipFree(d_mask);

    return 0;
}
