#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define TILE_SIZE 16

typedef long long ll;

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <filesystem>
#include "stb/stb_image.h"
#include "stb/stb_image_write.h"
using namespace std;

// Kernel function for 3D convolution
__global__ void inputTileConvolutionKernel(const unsigned char *inputImages, unsigned char *outputImages, const float *mask, int width, int height, int channels, int maskSize, int batchSize) {
    // Shared memory for input tile, the size of the tile is (TILE_SIZE + maskSize - 1)^2 passed as an argument
    extern __shared__ float tile[];

    // Mask Radius
    int maskRadius = maskSize / 2;

    // Tile dimension accounting for the mask radius
    int tileDim = TILE_SIZE + 2 * maskRadius;

    // Calculate the output image coordinates (outCol, outRow) inside the tile padded with mask radius
    int outCol = blockIdx.x * TILE_SIZE + threadIdx.x;
    int outRow = blockIdx.y * TILE_SIZE + threadIdx.y;

    // Calculate the input image coordinates (inCol, inRow) after applying the mask radius offset to the output image coordinates
    int inCol = outCol - maskRadius;
    int inRow = outRow - maskRadius;

    // Calculate the batch index based on the block index
    int batchIndex = blockIdx.z * blockDim.z + threadIdx.z;

    // Calculate the tile index
    int tileIndex = threadIdx.y * tileDim + threadIdx.x;
    // Calculate the input image index based on the batch index, input row, input column, and channel
    int imgIndex = (batchIndex * height * width + inRow * width + inCol) * channels;
    // Check if the input image pixel is valid (Boundary check)
    bool validPixel = (inCol >= 0 && inCol < width && inRow >= 0 && inRow < height);

    // Load the valid input image pixels into the tile (Only RGB channels are considered) and fill the rest with 0
    tile[tileIndex] = validPixel ? (float)inputImages[imgIndex] + (float)inputImages[imgIndex + 1] + (float)inputImages[imgIndex + 2] : 0;

    // Synchronize threads to ensure all threads have loaded the input image pixels into the tile
    __syncthreads();

    // Boundary check for the output image coordinates, batch index, and thread index
    if (outCol < width && outRow < height && batchIndex < batchSize && threadIdx.x < TILE_SIZE && threadIdx.y < TILE_SIZE) {
        // Initialize sum to 0
        float sum = 0.0;
        // Iterate over the mask
        for (int i = 0; i < maskSize; i++)
            for (int j = 0; j < maskSize; j++)
                // Multiply the mask element with the corresponding input image pixel value and add it to the sum
                sum += mask[i * maskSize + j] * tile[(threadIdx.y + i) * tileDim + threadIdx.x + j];

        // Clip the sum to the range [0, 255]
        sum = sum < 0 ? 0 : sum;
        sum = sum > 255 ? 255 : sum;
        // Store the result in the output image after linearizing the 2D output image to 1D
        outputImages[batchIndex * height * width + outRow * width + outCol] = (unsigned char)sum;
    }

    // Synchronize threads to ensure all threads have completed the convolution operation
    __syncthreads();
}

int main(int argc, char **argv) {
    // Check for the correct number of arguments
    if (argc != 5) {
        printf("Usage: %s input_folder output_folder batch_size mask_file\n", argv[0]);
        return 1;
    }

    // Read command line arguments
    string inputFolder = argv[1];
    string outputFolder = argv[2];
    int batchSize = atoi(argv[3]);
    char* maskFile = argv[4];

    // Print the command line arguments
    printf("Input folder: %s\n", inputFolder.c_str());
    printf("Output folder: %s\n", outputFolder.c_str());
    printf("Batch size: %d\n", batchSize);
    printf("Mask file: %s\n", maskFile);
    printf("---------------------------------------------\n");

    // Check if the input folder exists
    if (!filesystem::exists(inputFolder)) {
        printf("Error: Input folder does not exist\n");
        return 1;
    }

    // Check if the mask file exists
    if (!filesystem::exists(maskFile)) {
        printf("Error: Mask file does not exist\n");
        return 1;
    }

    // Find the number of images in the input folder
    int numImages = 0;
    for (const auto &entry: filesystem::directory_iterator(inputFolder)) numImages++;
    printf("Number of images: %d\n", numImages);

    // Check if the number of images is less than the batch size
    if (numImages < batchSize) {
        printf("Error: Number of images is less than the batch size\n");
        return 1;
    }

    // Get the width, height, and number of channels of the first image in the input folder
    int width, height, channels;
    string firstImage = filesystem::directory_iterator(inputFolder)->path().string();
    unsigned char *image = stbi_load(firstImage.c_str(), &width, &height, &channels, STBI_rgb);
    if (image == NULL) {
        printf("Error: Failed to load image\n");
        return 1;
    }

    // Print the width, height, and number of channels of the image
    printf("Image width: %d\n", width);
    printf("Image height: %d\n", height);
    printf("Number of channels: %d\n", channels);

    // Calculate the size of the images
    int imgSize = width * height * channels;
    printf("Image size: %d\n", imgSize);
    printf("---------------------------------------------\n");

    // Allocate memory for input and output images
    unsigned char *inputImages = (unsigned char *)malloc(imgSize * batchSize * sizeof(unsigned char));
    unsigned char *outputImages = (unsigned char *)malloc(imgSize * batchSize * sizeof(unsigned char));

    // Read input images from the input folder and copy them to the inputImages array
    int i = 0;
    for (const auto &entry: filesystem::directory_iterator(inputFolder)) {
        string imagePath = entry.path().string();
        unsigned char *image = stbi_load(imagePath.c_str(), &width, &height, &channels, STBI_rgb);
        if (image == NULL) {
            printf("Error: Failed to load image\n");
            return 1;
        }
        memcpy(inputImages + i * imgSize, image, imgSize * sizeof(unsigned char));
        stbi_image_free(image);
        i++;
        if (i == batchSize) break;
    }
    
    // Read convolution mask from the provided text file
    FILE *fp = fopen(maskFile, "r");
    if (fp == NULL) {
        printf("Error: Unable to open mask file\n");
        return 1;
    }
    int maskSize;
    fscanf(fp, "%d", &maskSize);
    printf("Mask size: %d\n", maskSize);
    float *mask = (float *)malloc(maskSize * maskSize * sizeof(float));
    if (mask == NULL) {
        printf("Error: Memory allocation failed for mask\n");
        fclose(fp);
        return 1;
    }
    for (int i = 0; i < maskSize * maskSize; i++) {
        fscanf(fp, "%f", &mask[i]);
    }
    fclose(fp);

    // Print the mask
    printf("Mask:\n");
    for (int i = 0; i < maskSize; i++) {
        for (int j = 0; j < maskSize; j++) {
            printf("%f ", mask[i * maskSize + j]);
        }
        printf("\n");
    }
    printf("---------------------------------------------\n");

    // Allocate memory on GPU for input images, output images, and mask
    unsigned char *d_inputImages, *d_outputImages;
    float *d_mask;
    hipMalloc(&d_inputImages, imgSize * batchSize * sizeof(unsigned char));
    hipMalloc(&d_outputImages, imgSize * batchSize * sizeof(unsigned char));
    hipMalloc(&d_mask, maskSize * maskSize * sizeof(float));

    // Copy input images and mask from host to device
    hipMemcpy(d_mask, mask, maskSize * maskSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_inputImages, inputImages, imgSize * batchSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Calculate grid and block dimensions
    dim3 block(TILE_SIZE + maskSize - 1, TILE_SIZE + maskSize - 1, 1);
    dim3 grid((width + block.x - 1) / TILE_SIZE, (height + block.y - 1) / TILE_SIZE, (batchSize + block.z - 1) / block.z);

    // Calculate shared memory size
    int sharedMemorySize = pow(TILE_SIZE + maskSize - 1, 2) * sizeof(float);

    // Launch kernel for all images in the batch
    inputTileConvolutionKernel<<<grid, block, sharedMemorySize>>>(d_inputImages, d_outputImages, d_mask, width, height, channels, maskSize, batchSize);

    // Copy results (output images) from device to host
    hipMemcpy(outputImages, d_outputImages, imgSize * batchSize * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Create output directory if it does not exist
    if (!filesystem::exists(outputFolder)) {
        filesystem::create_directory(outputFolder);
    }

    // Save output images to the specified output folder
    for (int i = 0; i < batchSize; i++) {
        string outputImagePath = outputFolder + "/" + to_string(i) + ".jpg";
        stbi_write_jpg(outputImagePath.c_str(), width, height, 1, outputImages + i * width * height, width);
    }
    printf("Output images saved to %s\n", outputFolder.c_str());

    // Free memory
    free(inputImages);
    free(outputImages);
    free(mask);
    hipFree(d_inputImages);
    hipFree(d_outputImages);
    hipFree(d_mask);

    return 0;
}
